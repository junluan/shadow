#include "allocator.hpp"

#include "external.hpp"

namespace Shadow {

class GPUAllocator : public Allocator {
 public:
  DeviceType device_type() const override { return DeviceType::kGPU; }

  void* malloc(size_t size, const void* host_ptr) const override {
    void* ptr;
    CUDA_CHECK(hipMalloc(&ptr, size));
    if (host_ptr != nullptr) {
      write(size, host_ptr, ptr);
    }
    return ptr;
  }

  void read(size_t size, const void* src, void* dst) const override {
    if (cuda_stream_ == nullptr) {
      CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
    } else {
      CUDA_CHECK(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost,
                                 cuda_stream_));
    }
  }

  void write(size_t size, const void* src, void* dst) const override {
    if (cuda_stream_ == nullptr) {
      CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
    } else {
      CUDA_CHECK(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice,
                                 cuda_stream_));
    }
  }

  void copy(size_t size, const void* src, void* dst) const override {
    if (cuda_stream_ == nullptr) {
      CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
    } else {
      CUDA_CHECK(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToDevice,
                                 cuda_stream_));
    }
  }

  void free(void* ptr) const override { CUDA_CHECK(hipFree(ptr)); }

  void set_stream(void* stream) override {
    cuda_stream_ = hipStream_t(stream);
  }

 private:
  hipStream_t cuda_stream_ = nullptr;
};

template <>
std::shared_ptr<Allocator> GetAllocator<DeviceType::kGPU>() {
  return std::make_shared<GPUAllocator>();
}

}  // namespace Shadow
