#include "hip/hip_runtime.h"
#include "decode_box_op.hpp"

namespace Shadow {

namespace Vision {

template <typename T>
__device__ void decode(const T *encode_box, const T *prior_box,
                       const T *prior_var, T *decode_box) {
  T prior_w = prior_box[2] - prior_box[0];
  T prior_h = prior_box[3] - prior_box[1];
  T prior_c_x = (prior_box[0] + prior_box[2]) / 2;
  T prior_c_y = (prior_box[1] + prior_box[3]) / 2;

  T decode_box_c_x = prior_var[0] * encode_box[0] * prior_w + prior_c_x;
  T decode_box_c_y = prior_var[1] * encode_box[1] * prior_h + prior_c_y;
  T decode_box_w = expf(prior_var[2] * encode_box[2]) * prior_w;
  T decode_box_h = expf(prior_var[3] * encode_box[3]) * prior_h;

  decode_box[0] = decode_box_c_x - decode_box_w / 2;
  decode_box[1] = decode_box_c_y - decode_box_h / 2;
  decode_box[2] = decode_box_c_x + decode_box_w / 2;
  decode_box[3] = decode_box_c_y + decode_box_h / 2;

  decode_box[0] = max(min(decode_box[0], T(1)), T(0));
  decode_box[1] = max(min(decode_box[1], T(1)), T(0));
  decode_box[2] = max(min(decode_box[2], T(1)), T(0));
  decode_box[3] = max(min(decode_box[3], T(1)), T(0));
}

template <typename T>
__global__ void KernelDecodeSSDBoxes(int count, const T *mbox_loc,
                                     const T *mbox_conf, const T *mbox_priorbox,
                                     int num_priors, int num_classes,
                                     T *decode_box) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int prior_index = globalid % num_priors;

    const T *prior_box = mbox_priorbox + prior_index * 4;
    const T *prior_var = mbox_priorbox + (num_priors + prior_index) * 4;

    mbox_loc += globalid * 4, mbox_conf += globalid * num_classes;
    decode_box += globalid * 6;

    decode<T>(mbox_loc, prior_box, prior_var, decode_box + 2);

    int max_index = -1;
    T max_score = -FLT_MAX;
    for (int c = 0; c < num_classes; ++c) {
      T score = mbox_conf[c];
      if (score > max_score) {
        max_index = c;
        max_score = score;
      }
    }
    decode_box[0] = max_index;
    decode_box[1] = max_score;
  }
}

template <typename T>
void DecodeSSDBoxes(const T *mbox_loc, const T *mbox_conf,
                    const T *mbox_priorbox, int batch, int num_priors,
                    int num_classes, T *decode_box, Context *context) {
  int count = batch * num_priors;
  KernelDecodeSSDBoxes<T><<<GetBlocks(count), NumThreads, 0,
                            hipStream_t(context->hip_stream())>>>(
      count, mbox_loc, mbox_conf, mbox_priorbox, num_priors, num_classes,
      decode_box);
  CUDA_CHECK(hipPeekAtLastError());
}

template void DecodeSSDBoxes(const float *, const float *, const float *, int,
                             int, int, float *, Context *);

template <typename T>
__global__ void KernelDecodeRefineDetBoxes(
    int count, const T *odm_loc, const T *odm_conf, const T *arm_priorbox,
    const T *arm_conf, const T *arm_loc, int num_priors, int num_classes,
    int background_label_id, float objectness_score, T *decode_box) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int prior_index = globalid % num_priors;

    const T *prior_box = arm_priorbox + prior_index * 4;
    const T *prior_var = arm_priorbox + (num_priors + prior_index) * 4;

    odm_loc += globalid * 4, odm_conf += globalid * num_classes;
    arm_conf += globalid * 2, arm_loc += globalid * 4;
    decode_box += globalid * 6;

    decode<T>(arm_loc, prior_box, prior_var, decode_box + 2);
    decode<T>(odm_loc, decode_box + 2, prior_var, decode_box + 2);

    if (arm_conf[1] < objectness_score) {
      decode_box[0] = background_label_id;
      decode_box[1] = 1;
    } else {
      int max_index = -1;
      T max_score = -FLT_MAX;
      for (int c = 0; c < num_classes; ++c) {
        T score = odm_conf[c];
        if (score > max_score) {
          max_index = c;
          max_score = score;
        }
      }
      decode_box[0] = max_index;
      decode_box[1] = max_score;
    }
  }
}

template <typename T>
void DecodeRefineDetBoxes(const T *odm_loc, const T *odm_conf,
                          const T *arm_priorbox, const T *arm_conf,
                          const T *arm_loc, int batch, int num_priors,
                          int num_classes, int background_label_id,
                          float objectness_score, T *decode_box,
                          Context *context) {
  int count = batch * num_priors;
  KernelDecodeRefineDetBoxes<T><<<GetBlocks(count), NumThreads, 0,
                                  hipStream_t(context->hip_stream())>>>(
      count, odm_loc, odm_conf, arm_priorbox, arm_conf, arm_loc, num_priors,
      num_classes, background_label_id, objectness_score, decode_box);
  CUDA_CHECK(hipPeekAtLastError());
}

template void DecodeRefineDetBoxes(const float *, const float *, const float *,
                                   const float *, const float *, int, int, int,
                                   int, float, float *, Context *);

template <typename T>
__global__ void KernelDecodeYoloV3Boxes(int count, const T *in_data,
                                        const T *biases, int num_priors,
                                        int out_h, int out_w, int mask,
                                        int num_classes, T *decode_box) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int temp = globalid / mask;
    int km_out = globalid % mask;
    int w_out = temp % out_w;
    temp /= out_w;
    int h_out = temp % out_h;
    int b_out = temp / out_h;

    in_data += globalid * (4 + 1 + num_classes);
    decode_box +=
        (b_out * num_priors + (h_out * out_w + w_out) * mask + km_out) * 6;

    float x = (1.f / (1 + expf(-in_data[0])) + w_out) / out_w;
    float y = (1.f / (1 + expf(-in_data[1])) + h_out) / out_h;
    float w = expf(in_data[2]) * biases[2 * km_out];
    float h = expf(in_data[3]) * biases[2 * km_out + 1];

    int max_index = -1;
    auto max_score = -FLT_MAX;
    float scale = 1.f / (1 + expf(-in_data[4]));
    for (int c = 0; c < num_classes; ++c) {
      float score = scale * 1.f / (1 + expf(-in_data[5 + c]));
      if (score > max_score) {
        max_index = c;
        max_score = score;
      }
    }

    decode_box[0] = max_index;
    decode_box[1] = max_score;
    decode_box[2] = x;
    decode_box[3] = y;
    decode_box[4] = w;
    decode_box[5] = h;
  }
}

template <typename T>
void DecodeYoloV3Boxes(const T *in_data, const T *biases, int batch,
                       int num_priors, int out_h, int out_w, int mask,
                       int num_classes, T *decode_box, Context *context) {
  int count = batch * out_h * out_w * mask;
  KernelDecodeYoloV3Boxes<T><<<GetBlocks(count), NumThreads, 0,
                               hipStream_t(context->hip_stream())>>>(
      count, in_data, biases, num_priors, out_h, out_w, mask, num_classes,
      decode_box);
  CUDA_CHECK(hipPeekAtLastError());
}

template void DecodeYoloV3Boxes(const float *in_data, const float *biases,
                                int batch, int num_priors, int out_h, int out_w,
                                int mask, int num_classes, float *decode_box,
                                Context *context);

}  // namespace Vision

}  // namespace Shadow
