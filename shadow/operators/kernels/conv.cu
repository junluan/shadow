#include "hip/hip_runtime.h"
#include "conv.hpp"

namespace Shadow {

namespace Vision {

__global__ void KernelIm2Col(const float* in_data, int offset, int count,
                             int in_c, int in_h, int in_w, int kernel_size_h,
                             int kernel_size_w, int stride_h, int stride_w,
                             int pad_h, int pad_w, int dilation, int zero_point,
                             int out_h, int out_w, float* col_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int h_index = globalid / out_w;
    int h_col = h_index % out_h;
    int w_col = globalid % out_w;
    int c_im = h_index / out_h;
    int c_col = c_im * kernel_size_h * kernel_size_w;
    int h_offset = h_col * stride_h - pad_h;
    int w_offset = w_col * stride_w - pad_w;
    col_data += (c_col * out_h + h_col) * out_w + w_col;
    in_data += offset + (c_im * in_h + h_offset) * in_w + w_offset;
    for (int i = 0; i < kernel_size_h; ++i) {
      for (int j = 0; j < kernel_size_w; ++j) {
        int h_im = h_offset + i * dilation;
        int w_im = w_offset + j * dilation;
        *col_data = (h_im >= 0 && w_im >= 0 && h_im < in_h && w_im < in_w)
                        ? in_data[i * dilation * in_w + j * dilation]
                        : static_cast<float>(zero_point);
        col_data += out_h * out_w;
      }
    }
  }
}

template <>
void Im2Col<DeviceType::kGPU, float>(const float* in_data,
                                     const VecInt& in_shape, int offset,
                                     int kernel_size_h, int kernel_size_w,
                                     int stride_h, int stride_w, int pad_h,
                                     int pad_w, int dilation, int zero_point,
                                     const VecInt& out_shape, float* col_data,
                                     Context* context) {
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int out_h = out_shape[2], out_w = out_shape[3];
  int count = in_c * out_h * out_w;
  KernelIm2Col<<<GetBlocks(count), NumThreads, 0,
                 hipStream_t(context->stream())>>>(
      in_data, offset, count, in_c, in_h, in_w, kernel_size_h, kernel_size_w,
      stride_h, stride_w, pad_h, pad_w, dilation, zero_point, out_h, out_w,
      col_data);
  CUDA_CHECK(hipPeekAtLastError());
}

__global__ void KernelDepthwise(
    const float* in_data, int count, const float* weight_data,
    const float* bias_data, int in_c, int in_h, int in_w, int out_h, int out_w,
    int kernel_size_h, int kernel_size_w, int stride_h, int stride_w, int pad_h,
    int pad_w, int dilation, bool bias_term, float* out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int w = globalid % out_w;
    int h = (globalid / out_w) % out_h;
    int c = (globalid / out_w / out_h) % in_c;
    int n = globalid / out_w / out_h / in_c;

    const auto* in_offset_data = in_data + (n * in_c + c) * in_h * in_w;
    const auto* weight_offset_data =
        weight_data + c * kernel_size_h * kernel_size_w;

    double sum_val = 0;
    for (int kh = 0; kh < kernel_size_h; ++kh) {
      for (int kw = 0; kw < kernel_size_w; ++kw) {
        int h_in = h * stride_h - pad_h + kh * dilation;
        int w_in = w * stride_w - pad_w + kw * dilation;
        if (h_in >= 0 && h_in < in_h && w_in >= 0 && w_in < in_w) {
          sum_val += in_offset_data[h_in * in_w + w_in] * *weight_offset_data;
        }
        weight_offset_data++;
      }
    }
    if (bias_term) {
      sum_val += bias_data[c];
    }

    out_data[globalid] = static_cast<float>(sum_val);
  }
}

template <>
void Depthwise<DeviceType::kGPU, float>(
    const float* in_data, const VecInt& in_shape, const float* weight_data,
    const float* bias_data, int kernel_size_h, int kernel_size_w, int stride_h,
    int stride_w, int pad_h, int pad_w, int dilation, bool bias_term,
    const VecInt& out_shape, float* out_data, Context* context) {
  int batch = in_shape[0];
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int out_h = out_shape[2], out_w = out_shape[3];
  int count = batch * in_c * out_h * out_w;
  KernelDepthwise<<<GetBlocks(count), NumThreads, 0,
                    hipStream_t(context->stream())>>>(
      in_data, count, weight_data, bias_data, in_c, in_h, in_w, out_h, out_w,
      kernel_size_h, kernel_size_w, stride_h, stride_w, pad_h, pad_w, dilation,
      bias_term, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

}  // namespace Vision

}  // namespace Shadow

namespace Shadow {

REGISTER_OP_KERNEL_DEFAULT(ConvGPU, ConvKernelDefault<DeviceType::kGPU>);

#if defined(USE_CUDNN)

class ConvKernelCUDNN : public ConvKernel {
 public:
  ConvKernelCUDNN() {
    cudnn::createConvolutionDesc<float>(&conv_desc_);
    cudnn::createTensorDesc<float>(&in_desc_);
    cudnn::createTensorDesc<float>(&out_desc_);
    cudnn::createFilterDesc<float>(&weight_desc_);
    cudnn::createTensorDesc<float>(&bias_desc_);
    cudnn::createActivationDesc<float>(&activate_desc_);
  }
  ~ConvKernelCUDNN() override {
    if (conv_desc_ != nullptr) {
      hipdnnDestroyConvolutionDescriptor(conv_desc_);
      conv_desc_ = nullptr;
    }
    if (in_desc_ != nullptr) {
      hipdnnDestroyTensorDescriptor(in_desc_);
      in_desc_ = nullptr;
    }
    if (out_desc_ != nullptr) {
      hipdnnDestroyTensorDescriptor(out_desc_);
      out_desc_ = nullptr;
    }
    if (weight_desc_ != nullptr) {
      hipdnnDestroyFilterDescriptor(weight_desc_);
      weight_desc_ = nullptr;
    }
    if (bias_desc_ != nullptr) {
      hipdnnDestroyTensorDescriptor(bias_desc_);
      bias_desc_ = nullptr;
    }
    if (activate_desc_ != nullptr) {
      hipdnnDestroyActivationDescriptor(activate_desc_);
      activate_desc_ = nullptr;
    }
  }

  void Run(const std::shared_ptr<Blob>& input,
           const std::shared_ptr<Blob>& weight,
           const std::shared_ptr<Blob>& bias, std::shared_ptr<Blob>& output,
           Workspace* ws, int num_output, int kernel_size_h, int kernel_size_w,
           int stride_h, int stride_w, int pad_h, int pad_w, int dilation,
           int group, bool bias_term, int activate_type) override {
    int batch = input->shape(0), in_c = input->shape(1), in_h = input->shape(2),
        in_w = input->shape(3);
    int out_h = output->shape(2), out_w = output->shape(3);

    cudnn::setConvolution2dDesc<float>(&conv_desc_, pad_h, pad_w, stride_h,
                                       stride_w, dilation, dilation, group);
    cudnn::setTensor4dDesc<float>(&in_desc_, batch, in_c, in_h, in_w);
    cudnn::setTensor4dDesc<float>(&out_desc_, batch, num_output, out_h, out_w);
    cudnn::setFilter4dDesc<float>(&weight_desc_, num_output, in_c / group,
                                  kernel_size_h, kernel_size_w);
    if (bias_term) {
      cudnn::setTensor4dDesc<float>(&bias_desc_, 1, num_output, 1, 1);
    }
    if (activate_type == 1) {
      cudnn::setActivationDesc<float>(&activate_desc_, activate_type, 0);
    }

    size_t workspace_limit_bytes = group == 1 ? 64 * 1024 * 1024 : 0;

    CUDNN_CHECK(hipdnnGetConvolutionForwardAlgorithm(
        hipdnnHandle_t(ws->Ctx()->cudnn_handle()), in_desc_, weight_desc_,
        conv_desc_, out_desc_, HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
        workspace_limit_bytes, &fwd_algo_));

    size_t workspace_fwd_size = 0;

    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(
        hipdnnHandle_t(ws->Ctx()->cudnn_handle()), in_desc_, weight_desc_,
        conv_desc_, out_desc_, fwd_algo_, &workspace_fwd_size));

    std::shared_ptr<Blob> workspace = nullptr;
    const void* workspace_ptr = nullptr;
    if (workspace_fwd_size > 0) {
      ws->GrowTempBuffer(workspace_fwd_size);
      workspace = ws->CreateTempBlob({static_cast<int>(workspace_fwd_size)},
                                     DataType::kU8);
      workspace_ptr = workspace->data<unsigned char>();
    }

    CUDNN_CHECK(hipdnnConvolutionForward(
        hipdnnHandle_t(ws->Ctx()->cudnn_handle()), cudnn::dataType<float>::one,
        in_desc_, input->data<float>(), weight_desc_, weight->data<float>(),
        conv_desc_, fwd_algo_, const_cast<void*>(workspace_ptr),
        workspace_fwd_size, cudnn::dataType<float>::zero, out_desc_,
        output->mutable_data<float>()));

    if (bias_term) {
      CUDNN_CHECK(hipdnnAddTensor(
          hipdnnHandle_t(ws->Ctx()->cudnn_handle()), cudnn::dataType<float>::one,
          bias_desc_, bias->data<float>(), cudnn::dataType<float>::one,
          out_desc_, output->mutable_data<float>()));
    }

    if (activate_type == 1) {
      CUDNN_CHECK(hipdnnActivationForward(
          hipdnnHandle_t(ws->Ctx()->cudnn_handle()), activate_desc_,
          cudnn::dataType<float>::one, out_desc_, output->data<float>(),
          cudnn::dataType<float>::zero, out_desc_,
          output->mutable_data<float>()));
    }
  }

  DeviceType device_type() const override { return DeviceType::kGPU; }

  std::string kernel_type() const override { return "CUDNN"; }

 private:
  hipdnnConvolutionFwdAlgo_t fwd_algo_ =
      HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;

  hipdnnConvolutionDescriptor_t conv_desc_ = nullptr;
  hipdnnTensorDescriptor_t in_desc_ = nullptr, out_desc_ = nullptr;
  hipdnnFilterDescriptor_t weight_desc_ = nullptr;
  hipdnnTensorDescriptor_t bias_desc_ = nullptr;

  hipdnnActivationDescriptor_t activate_desc_ = nullptr;
};

REGISTER_OP_KERNEL_CUDNN(ConvGPU, ConvKernelCUDNN);

#endif

}  // namespace Shadow
