#include "hip/hip_runtime.h"
#include "normalize.hpp"

namespace Shadow {

namespace Vision {

__global__ void KernelLpNorm(const float* in_data, int val_count, int dim,
                             int inner_num, float p, float* val_data) {
  CUDA_KERNEL_LOOP(globalid, val_count) {
    int n = globalid / inner_num, s = globalid % inner_num;
    const auto* in_data_offset = in_data + n * dim * inner_num + s;
    double val = 0;
    if (p == 1) {
      for (int c = 0; c < dim; ++c, in_data_offset += inner_num) {
        val += fabsf(*in_data_offset);
      }
      val_data[globalid] = static_cast<float>(val);
    } else if (p == 2) {
      for (int c = 0; c < dim; ++c, in_data_offset += inner_num) {
        auto abs_data = fabsf(*in_data_offset);
        val += abs_data * abs_data;
      }
      val_data[globalid] = sqrtf(static_cast<float>(val));
    } else {
      for (int c = 0; c < dim; ++c, in_data_offset += inner_num) {
        val += powf(fabsf(*in_data_offset), p);
      }
      val_data[globalid] = powf(static_cast<float>(val), 1.f / p);
    }
  }
}

__global__ void KernelDivLpNorm(const float* in_data, const float* val_data,
                                int count, int dim, int inner_num, float eps,
                                float* out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int n = globalid / dim / inner_num, s = globalid % inner_num;
    out_data[globalid] =
        in_data[globalid] / fmaxf(val_data[n * inner_num + s], eps);
  }
}

template <>
void Normalize<DeviceType::kGPU, float>(const float* in_data, int outer_num,
                                        int dim, int inner_num, float* val_data,
                                        float p, float eps, float* out_data,
                                        Context* context) {
  int val_count = outer_num * inner_num, count = val_count * dim;
  KernelLpNorm<<<GetBlocks(val_count), NumThreads, 0,
                 hipStream_t(context->hip_stream())>>>(
      in_data, val_count, dim, inner_num, p, val_data);
  KernelDivLpNorm<<<GetBlocks(count), NumThreads, 0,
                    hipStream_t(context->hip_stream())>>>(
      in_data, val_data, count, dim, inner_num, eps, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

}  // namespace Vision

}  // namespace Shadow

namespace Shadow {

REGISTER_OP_KERNEL_DEFAULT(NormalizeGPU,
                           NormalizeKernelDefault<DeviceType::kGPU>);

}  // namespace Shadow
