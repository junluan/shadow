#include "hip/hip_runtime.h"
#include "roi_align.hpp"

namespace Shadow {

namespace Vision {

__device__ float roi_align_bilinear(const float* data, float x, float y,
                                    int width, int height) {
  if (x < -1.0 || x > width || y < -1.0 || y > height) {
    return 0;
  }

  x = fmaxf(x, 0.f), y = fmaxf(y, 0.f);

  auto h_low = static_cast<int>(floorf(y));
  auto w_low = static_cast<int>(floorf(x));
  int h_high = h_low + 1, w_high = w_low + 1;

  if (h_low >= height - 1) {
    h_high = h_low = height - 1;
    y = static_cast<float>(h_low);
  }

  if (w_low >= width - 1) {
    w_high = w_low = width - 1;
    x = static_cast<float>(w_low);
  }

  float lh = y - h_low, lw = x - w_low;
  float hh = 1 - lh, hw = 1 - lw;
  float v1 = data[h_low * width + w_low];
  float v2 = data[h_low * width + w_high];
  float v3 = data[h_high * width + w_low];
  float v4 = data[h_high * width + w_high];
  float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  return w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4;
}

__global__ void KernelPOIAlign(const float* in_data, int count,
                               const float* roi_data, int in_c, int in_h,
                               int in_w, int pooled_h, int pooled_w,
                               float spatial_scale, int sampling_ratio,
                               bool align_corners, float* out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int pw = globalid % pooled_w;
    int ph = (globalid / pooled_w) % pooled_h;
    int c = (globalid / pooled_w / pooled_h) % in_c;
    int n = globalid / pooled_w / pooled_h / in_c;

    roi_data += n * 5;
    int roi_batch_id = static_cast<int>(roi_data[0]);

    float offset = align_corners ? 0.5f : 0.f;
    float roi_start_w = roi_data[1] * spatial_scale - offset;
    float roi_start_h = roi_data[2] * spatial_scale - offset;
    float roi_end_w = roi_data[3] * spatial_scale - offset;
    float roi_end_h = roi_data[4] * spatial_scale - offset;

    float roi_height = roi_end_h - roi_start_h;
    float roi_width = roi_end_w - roi_start_w;

    if (!align_corners) {
      roi_height = fmaxf(roi_height, 1.f);
      roi_width = fmaxf(roi_width, 1.f);
    }

    float bin_size_h = roi_height / pooled_h;
    float bin_size_w = roi_width / pooled_w;

    int roi_bin_grid_h = sampling_ratio > 0
                             ? sampling_ratio
                             : static_cast<int>(ceilf(bin_size_h));
    int roi_bin_grid_w = sampling_ratio > 0
                             ? sampling_ratio
                             : static_cast<int>(ceilf(bin_size_w));

    float grid_size = max(roi_bin_grid_h * roi_bin_grid_w, 1);

    in_data += (roi_batch_id * in_c + c) * in_h * in_w;

    double sum_val = 0;
    for (int h = 0; h < roi_bin_grid_h; ++h) {
      float y = roi_start_h + ph * bin_size_h +
                (h + 0.5f) * bin_size_h / roi_bin_grid_h;
      for (int w = 0; w < roi_bin_grid_w; ++w) {
        float x = roi_start_w + pw * bin_size_w +
                  (w + 0.5f) * bin_size_w / roi_bin_grid_w;
        sum_val += roi_align_bilinear(in_data, x, y, in_w, in_h);
      }
    }

    out_data[globalid] = static_cast<float>(sum_val / grid_size);
  }
}

template <>
void ROIAlign<DeviceType::kGPU, float>(
    const float* in_data, const VecInt& in_shape, const float* roi_data,
    int num_rois, int pooled_h, int pooled_w, float spatial_scale,
    int sampling_ratio, bool align_corners, float* out_data, Context* context) {
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int count = num_rois * in_c * pooled_h * pooled_w;
  KernelPOIAlign<<<GetBlocks(count), NumThreads, 0,
                   hipStream_t(context->stream())>>>(
      in_data, count, roi_data, in_c, in_h, in_w, pooled_h, pooled_w,
      spatial_scale, sampling_ratio, align_corners, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

}  // namespace Vision

}  // namespace Shadow

namespace Shadow {

REGISTER_OP_KERNEL_DEFAULT(ROIAlignGPU,
                           ROIAlignKernelDefault<DeviceType::kGPU>);

}  // namespace Shadow
