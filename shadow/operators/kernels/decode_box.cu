#include "hip/hip_runtime.h"
#include "decode_box.hpp"

namespace Shadow {

namespace Vision {

__device__ void decode(const float* encode_box, const float* prior_box,
                       const float* prior_var, float* decode_box) {
  auto prior_w = prior_box[2] - prior_box[0];
  auto prior_h = prior_box[3] - prior_box[1];
  auto prior_c_x = (prior_box[0] + prior_box[2]) / 2.f;
  auto prior_c_y = (prior_box[1] + prior_box[3]) / 2.f;

  auto decode_box_c_x = prior_var[0] * encode_box[0] * prior_w + prior_c_x;
  auto decode_box_c_y = prior_var[1] * encode_box[1] * prior_h + prior_c_y;
  auto decode_box_w = expf(prior_var[2] * encode_box[2]) * prior_w;
  auto decode_box_h = expf(prior_var[3] * encode_box[3]) * prior_h;

  decode_box[0] = decode_box_c_x - decode_box_w / 2.f;
  decode_box[1] = decode_box_c_y - decode_box_h / 2.f;
  decode_box[2] = decode_box_c_x + decode_box_w / 2.f;
  decode_box[3] = decode_box_c_y + decode_box_h / 2.f;

  decode_box[0] = fmaxf(fminf(decode_box[0], 1.f), 0.f);
  decode_box[1] = fmaxf(fminf(decode_box[1], 1.f), 0.f);
  decode_box[2] = fmaxf(fminf(decode_box[2], 1.f), 0.f);
  decode_box[3] = fmaxf(fminf(decode_box[3], 1.f), 0.f);
}

__global__ void KernelDecodeSSDBoxes(int count, const float* mbox_loc,
                                     const float* mbox_conf,
                                     const float* mbox_priorbox, int num_priors,
                                     int num_classes, bool output_max_score,
                                     float* decode_box) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int prior_index = globalid % num_priors;

    const auto* prior_box = mbox_priorbox + prior_index * 4;
    const auto* prior_var = mbox_priorbox + (num_priors + prior_index) * 4;

    const auto* mbox_loc_offset = mbox_loc + globalid * 4;
    const auto* mbox_conf_offset = mbox_conf + globalid * num_classes;

    auto* box =
        decode_box + globalid * (output_max_score ? 6 : (4 + num_classes));

    if (output_max_score) {
      decode(mbox_loc_offset, prior_box, prior_var, box + 2);
      int max_index = -1;
      auto max_score = -FLT_MAX;
      for (int c = 0; c < num_classes; ++c) {
        auto score = mbox_conf_offset[c];
        if (score > max_score) {
          max_index = c;
          max_score = score;
        }
      }
      box[0] = max_index, box[1] = max_score;
    } else {
      decode(mbox_loc_offset, prior_box, prior_var, box);
      for (int c = 0; c < num_classes; ++c) {
        box[4 + c] = mbox_conf_offset[c];
      }
    }
  }
}

template <>
void DecodeSSDBoxes<DeviceType::kGPU, float>(
    const float* mbox_loc, const float* mbox_conf, const float* mbox_priorbox,
    int batch, int num_priors, int num_classes, bool output_max_score,
    float* decode_box, Context* context) {
  int count = batch * num_priors;
  KernelDecodeSSDBoxes<<<GetBlocks(count), NumThreads, 0,
                         hipStream_t(context->stream())>>>(
      count, mbox_loc, mbox_conf, mbox_priorbox, num_priors, num_classes,
      output_max_score, decode_box);
  CUDA_CHECK(hipPeekAtLastError());
}

__global__ void KernelDecodeRefineDetBoxes(
    int count, const float* odm_loc, const float* odm_conf,
    const float* arm_priorbox, const float* arm_conf, const float* arm_loc,
    int num_priors, int num_classes, int background_label_id,
    float objectness_score, bool output_max_score, float* decode_box) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int prior_index = globalid % num_priors;

    const auto* prior_box = arm_priorbox + prior_index * 4;
    const auto* prior_var = arm_priorbox + (num_priors + prior_index) * 4;

    const auto* odm_loc_offset = odm_loc + globalid * 4;
    const auto* odm_conf_offset = odm_conf + globalid * num_classes;
    const auto* arm_conf_offset = arm_conf + globalid * 2;
    const auto* arm_loc_offset = arm_loc + globalid * 4;

    auto* box =
        decode_box + globalid * (output_max_score ? 6 : (4 + num_classes));

    bool is_background = arm_conf_offset[1] < objectness_score;

    if (output_max_score) {
      decode(arm_loc_offset, prior_box, prior_var, box + 2);
      decode(odm_loc_offset, box + 2, prior_var, box + 2);
      if (is_background) {
        box[0] = background_label_id, box[1] = 1;
      } else {
        int max_index = -1;
        auto max_score = -FLT_MAX;
        for (int c = 0; c < num_classes; ++c) {
          auto score = odm_conf_offset[c];
          if (score > max_score) {
            max_index = c;
            max_score = score;
          }
        }
        box[0] = max_index, box[1] = max_score;
      }
    } else {
      decode(arm_loc_offset, prior_box, prior_var, box);
      decode(odm_loc_offset, box, prior_var, box);
      for (int c = 0; c < num_classes; ++c) {
        box[4 + c] = is_background ? 0 : odm_conf_offset[c];
      }
      if (is_background) {
        box[4 + background_label_id] = 1;
      }
    }
  }
}

template <>
void DecodeRefineDetBoxes<DeviceType::kGPU, float>(
    const float* odm_loc, const float* odm_conf, const float* arm_priorbox,
    const float* arm_conf, const float* arm_loc, int batch, int num_priors,
    int num_classes, int background_label_id, float objectness_score,
    bool output_max_score, float* decode_box, Context* context) {
  int count = batch * num_priors;
  KernelDecodeRefineDetBoxes<<<GetBlocks(count), NumThreads, 0,
                               hipStream_t(context->stream())>>>(
      count, odm_loc, odm_conf, arm_priorbox, arm_conf, arm_loc, num_priors,
      num_classes, background_label_id, objectness_score, output_max_score,
      decode_box);
  CUDA_CHECK(hipPeekAtLastError());
}

__global__ void KernelDecodeYoloV3Boxes(int count, const float* in_data,
                                        const float* biases, int num_priors,
                                        int out_h, int out_w, int mask,
                                        int num_classes, bool output_max_score,
                                        float* decode_box) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int temp = globalid / mask;
    int km_out = globalid % mask;
    int w_out = temp % out_w;
    temp /= out_w;
    int h_out = temp % out_h;
    int b_out = temp / out_h;

    const auto* in_data_offset = in_data + globalid * (4 + 1 + num_classes);

    float x = (1.f / (1 + expf(-in_data_offset[0])) + w_out) / out_w;
    float y = (1.f / (1 + expf(-in_data_offset[1])) + h_out) / out_h;
    float w = expf(in_data_offset[2]) * biases[2 * km_out];
    float h = expf(in_data_offset[3]) * biases[2 * km_out + 1];

    float scale = 1.f / (1 + expf(-in_data_offset[4]));

    int out_num =
        (b_out * num_priors + (h_out * out_w + w_out) * mask + km_out);

    auto* box =
        decode_box + out_num * (output_max_score ? 6 : (4 + num_classes));

    if (output_max_score) {
      int max_index = -1;
      auto max_score = -FLT_MAX;
      for (int c = 0; c < num_classes; ++c) {
        float score = scale / (1 + expf(-in_data_offset[5 + c]));
        if (score > max_score) {
          max_index = c;
          max_score = score;
        }
      }
      box[0] = max_index, box[1] = max_score;
      box[2] = x, box[3] = y, box[4] = w, box[5] = h;
    } else {
      box[0] = x, box[1] = y, box[2] = w, box[3] = h;
      for (int c = 0; c < num_classes; ++c) {
        box[4 + c] = scale / (1 + expf(-in_data_offset[5 + c]));
      }
    }
  }
}

template <>
void DecodeYoloV3Boxes<DeviceType::kGPU, float>(
    const float* in_data, const float* biases, int batch, int num_priors,
    int out_h, int out_w, int mask, int num_classes, bool output_max_score,
    float* decode_box, Context* context) {
  int count = batch * out_h * out_w * mask;
  KernelDecodeYoloV3Boxes<<<GetBlocks(count), NumThreads, 0,
                            hipStream_t(context->stream())>>>(
      count, in_data, biases, num_priors, out_h, out_w, mask, num_classes,
      output_max_score, decode_box);
  CUDA_CHECK(hipPeekAtLastError());
}

}  // namespace Vision

}  // namespace Shadow

namespace Shadow {

REGISTER_OP_KERNEL_DEFAULT(DecodeBoxGPU,
                           DecodeBoxKernelDefault<DeviceType::kGPU>);

}  // namespace Shadow
